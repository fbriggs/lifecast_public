#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/** @file   mlp-learning-an-image.cu
 *  @author Thomas Müller, NVIDIA
 *  @brief  Sample application that uses the tiny cuda nn framework to learn a
            2D function that represents an image.
 */

#include <tiny-cuda-nn/common_device.h>

#include <tiny-cuda-nn/config.h>

#include <stbi/stbi_wrapper.h>

#include <chrono>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <random>
#include <stdexcept>
#include <string>
#include <thread>
#include <vector>

using namespace tcnn;
using precision_t = network_precision_t;

GPUMemory<float> load_image(const std::string& filename, int& width, int& height) {
	// width * height * RGBA
	float* out = load_stbi(&width, &height, filename.c_str());

	GPUMemory<float> result(width * height * 4);
	result.copy_from_host(out);
	free(out); // release memory of image data

	return result;
}

template <typename T>
__global__ void to_ldr(const uint64_t num_elements, const uint32_t n_channels, const uint32_t stride, const T* __restrict__ in, uint8_t* __restrict__ out) {
	const uint64_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= num_elements) return;

	const uint64_t pixel = i / n_channels;
	const uint32_t channel = i - pixel * n_channels;

	out[i] = (uint8_t)(powf(fmaxf(fminf(in[pixel * stride + channel], 1.0f), 0.0f), 1.0f/2.2f) * 255.0f + 0.5f);
}

template <typename T>
void save_image(const T* image, int width, int height, int n_channels, int channel_stride, const std::string& filename) {
	GPUMemory<uint8_t> image_ldr(width * height * n_channels);
	linear_kernel(to_ldr<T>, 0, nullptr, width * height * n_channels, n_channels, channel_stride, image, image_ldr.data());

	std::vector<uint8_t> image_ldr_host(width * height * n_channels);
	CUDA_CHECK_THROW(hipMemcpy(image_ldr_host.data(), image_ldr.data(), image_ldr.size(), hipMemcpyDeviceToHost));

	save_stbi(image_ldr_host.data(), width, height, n_channels, filename.c_str());
}

template <uint32_t stride>
__global__ void eval_image(uint32_t n_elements, hipTextureObject_t texture, float* __restrict__ xs_and_ys, float* __restrict__ result) {
	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n_elements) return;

	uint32_t output_idx = i * stride;
	uint32_t input_idx = i * 2;

	float4 val = tex2D<float4>(texture, xs_and_ys[input_idx], xs_and_ys[input_idx+1]);
	result[output_idx + 0] = val.x;
	result[output_idx + 1] = val.y;
	result[output_idx + 2] = val.z;

	for (uint32_t i = 3; i < stride; ++i) {
		result[output_idx + i] = 1;
	}
}

int main(int argc, char* argv[]) {
	try {
		uint32_t compute_capability = cuda_compute_capability();
		if (compute_capability < MIN_GPU_ARCH) {
			std::cerr
				<< "Warning: Insufficient compute capability " << compute_capability << " detected. "
				<< "This program was compiled for >=" << MIN_GPU_ARCH << " and may thus behave unexpectedly." << std::endl;
		}

		if (argc < 2) {
			std::cout << "USAGE: " << argv[0] << " " << "path-to-image.jpg [path-to-optional-config.json]" << std::endl;
			std::cout << "Sample EXR files are provided in 'data/images'." << std::endl;
			return 0;
		}

		json config = {
			{"loss", {
				{"otype", "RelativeL2"}
			}},
			{"optimizer", {
				{"otype", "Adam"},
				// {"otype", "Shampoo"},
				{"learning_rate", 1e-2},
				{"beta1", 0.9f},
				{"beta2", 0.99f},
				{"l2_reg", 0.0f},
				// The following parameters are only used when the optimizer is "Shampoo".
				{"beta3", 0.9f},
				{"beta_shampoo", 0.0f},
				{"identity", 0.0001f},
				{"cg_on_momentum", false},
				{"frobenius_normalization", true},
			}},
			{"encoding", {
				{"otype", "OneBlob"},
				{"n_bins", 32},
			}},
			{"network", {
				{"otype", "FullyFusedMLP"},
				// {"otype", "CutlassMLP"},
				{"n_neurons", 64},
				{"n_hidden_layers", 4},
				{"activation", "ReLU"},
				{"output_activation", "None"},
			}},
		};

		if (argc >= 3) {
			std::cout << "Loading custom json config '" << argv[2] << "'." << std::endl;
			std::ifstream f{argv[2]};
			config = json::parse(f, nullptr, true, /*skip_comments=*/true);
		}

		// First step: load an image that we'd like to learn
		int width, height;
		GPUMemory<float> image = load_image(argv[1], width, height);

		// Second step: create a cuda texture out of this image. It'll be used to generate training data efficiently on the fly
		hipResourceDesc resDesc;
		memset(&resDesc, 0, sizeof(resDesc));
		resDesc.resType = hipResourceTypePitch2D;
		resDesc.res.pitch2D.devPtr = image.data();
		resDesc.res.pitch2D.desc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
		resDesc.res.pitch2D.width = width;
		resDesc.res.pitch2D.height = height;
		resDesc.res.pitch2D.pitchInBytes = width * 4 * sizeof(float);

		hipTextureDesc texDesc;
		memset(&texDesc, 0, sizeof(texDesc));
		texDesc.filterMode = hipFilterModeLinear;
		texDesc.normalizedCoords = true;
		texDesc.addressMode[0] = hipAddressModeClamp;
		texDesc.addressMode[1] = hipAddressModeClamp;
		texDesc.addressMode[2] = hipAddressModeClamp;

		hipTextureObject_t texture;
		CUDA_CHECK_THROW(hipCreateTextureObject(&texture, &resDesc, &texDesc, nullptr));

		// Third step: sample a reference image to dump to disk. Visual comparison of this reference image and the learned
		//             function will be eventually possible.

		int sampling_width = width;
		int sampling_height = height;

		// Uncomment to fix the resolution of the training task independent of input image
		// int sampling_width = 1024;
		// int sampling_height = 1024;

		uint32_t n_coords = sampling_width * sampling_height;
		uint32_t n_coords_padded = next_multiple(n_coords, BATCH_SIZE_GRANULARITY);

		GPUMemory<float> sampled_image(n_coords * 3);
		GPUMemory<float> xs_and_ys(n_coords_padded * 2);

		std::vector<float> host_xs_and_ys(n_coords * 2);
		for (int y = 0; y < sampling_height; ++y) {
			for (int x = 0; x < sampling_width; ++x) {
				int idx = (y * sampling_width + x) * 2;
				host_xs_and_ys[idx+0] = (float)(x + 0.5) / (float)sampling_width;
				host_xs_and_ys[idx+1] = (float)(y + 0.5) / (float)sampling_height;
			}
		}

		xs_and_ys.copy_from_host(host_xs_and_ys.data());

		linear_kernel(eval_image<3>, 0, nullptr, n_coords, texture, xs_and_ys.data(), sampled_image.data());

		save_image(sampled_image.data(), sampling_width, sampling_height, 3, 3, "reference.jpg");

		// Fourth step: train the model by sampling the above image and optimizing an error metric

		// Various constants for the network and optimization
		const uint32_t batch_size = 1 << 18;
		const uint32_t n_training_steps = argc >= 4 ? atoi(argv[3]) : 10000000;
		const uint32_t n_input_dims = 2; // 2-D image coordinate
		const uint32_t n_output_dims = 3; // RGB color

		hipStream_t inference_stream;
		CUDA_CHECK_THROW(hipStreamCreate(&inference_stream));
		hipStream_t training_stream = inference_stream;

		default_rng_t rng{1337};

		// Auxiliary matrices for training
		GPUMatrix<float> training_target(n_output_dims, batch_size);
		GPUMatrix<float> training_batch(n_input_dims, batch_size);

		// Auxiliary matrices for evaluation
		GPUMatrix<float> prediction(n_output_dims, n_coords_padded);
		GPUMatrix<float> inference_batch(xs_and_ys.data(), n_input_dims, n_coords_padded);

		json encoding_opts = config.value("encoding", json::object());
		json loss_opts = config.value("loss", json::object());
		json optimizer_opts = config.value("optimizer", json::object());
		json network_opts = config.value("network", json::object());

		std::shared_ptr<Loss<precision_t>> loss{create_loss<precision_t>(loss_opts)};
		std::shared_ptr<Optimizer<precision_t>> optimizer{create_optimizer<precision_t>(optimizer_opts)};
		std::shared_ptr<NetworkWithInputEncoding<precision_t>> network = std::make_shared<NetworkWithInputEncoding<precision_t>>(n_input_dims, n_output_dims, encoding_opts, network_opts);

		auto trainer = std::make_shared<Trainer<float, precision_t, precision_t>>(network, optimizer, loss);

		std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

		float tmp_loss = 0;
		uint32_t tmp_loss_counter = 0;

		std::cout << "Beginning optimization with " << n_training_steps << " training steps." << std::endl;

		uint32_t interval = 10;

		for (uint32_t i = 0; i < n_training_steps; ++i) {
			bool print_loss = i % interval == 0;
			bool visualize_learned_func = argc < 5 && i % interval == 0;

			// Compute reference values at random coordinates
			{
				generate_random_uniform<float>(training_stream, rng, batch_size * n_input_dims, training_batch.data());
				linear_kernel(eval_image<n_output_dims>, 0, training_stream, batch_size, texture, training_batch.data(), training_target.data());
			}

			// Training step
			{
				auto ctx = trainer->training_step(training_stream, training_batch, training_target);

				if (i % std::min(interval, (uint32_t)100) == 0) {
					tmp_loss += trainer->loss(training_stream, *ctx);
					++tmp_loss_counter;
				}
			}

			// Debug outputs
			{
				if (print_loss) {
					std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
					std::cout << "Step#" << i << ": " << "loss=" << tmp_loss/(float)tmp_loss_counter << " time=" << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "[µs]" << std::endl;

					tmp_loss = 0;
					tmp_loss_counter = 0;
				}

				if (visualize_learned_func) {
					network->inference(inference_stream, inference_batch, prediction);
					auto filename = fmt::format("{}.jpg", i);
					std::cout << "Writing '" << filename << "'... ";
					save_image(prediction.data(), sampling_width, sampling_height, 3, n_output_dims, filename);
					std::cout << "done." << std::endl;
				}

				// Don't count visualizing as part of timing
				// (assumes visualize_learned_pdf is only true when print_loss is true)
				if (print_loss) {
					begin = std::chrono::steady_clock::now();
				}
			}

			if (print_loss && i > 0 && interval < 1000) {
				interval *= 10;
			}
		}

		// Dump final image if a name was specified
		if (argc >= 5) {
			network->inference(inference_stream, inference_batch, prediction);
			save_image(prediction.data(), sampling_width, sampling_height, 3, n_output_dims, argv[4]);
		}

		free_all_gpu_memory_arenas();

		// If only the memory arenas pertaining to a single stream are to be freed, use
		//free_gpu_memory_arena(stream);
	} catch (const std::exception& e) {
		std::cout << "Uncaught exception: " << e.what() << std::endl;
	}

	return EXIT_SUCCESS;
}

