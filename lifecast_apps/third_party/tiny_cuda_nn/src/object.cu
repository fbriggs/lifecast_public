#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/** @file   object.cu
 *  @author Thomas Müller, NVIDIA
 *  @brief  API interface of a TCNN object
 */

#include <tiny-cuda-nn/common_device.h>
#include <tiny-cuda-nn/object.h>

namespace tcnn {

template <typename T>
__global__ void one_hot_batched_kernel(const uint32_t num_elements, const uint32_t width, const uint32_t one_hot_dim, T* out, float scale) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= num_elements) return;

	const uint32_t dim = i % width;
	out[i] = dim == one_hot_dim ? (T)scale : (T)0.0f;
}

template <typename T>
void one_hot_batched(hipStream_t stream, const uint32_t num_elements, const uint32_t width, const uint32_t one_hot_dim, T* out, float scale) {
	linear_kernel(one_hot_batched_kernel<T>, 0, stream, num_elements, width, one_hot_dim, out, scale);
}

template void one_hot_batched(hipStream_t stream, const uint32_t num_elements, const uint32_t width, const uint32_t one_hot_dim, float* out, float scale);
template void one_hot_batched(hipStream_t stream, const uint32_t num_elements, const uint32_t width, const uint32_t one_hot_dim, __half* out, float scale);

template <typename T>
void mult(hipStream_t stream, const uint32_t num_elements, T* inout, float factor) {
	linear_kernel(mult_scalar_kernel<T>, 0, stream, num_elements, inout, factor);
}

template void mult(hipStream_t stream, const uint32_t num_elements, float* inout, float factor);
template void mult(hipStream_t stream, const uint32_t num_elements, __half* inout, float factor);

template <typename T>
void trim_and_cast_from(hipStream_t stream, const MatrixLayout layout, const uint32_t num_elements, const uint32_t input_width, const uint32_t output_width, const T* in, float* out) {
	if (layout == RM) {
		linear_kernel(cast_from<T>, 0, stream, num_elements, in, out);
	} else {
		linear_kernel(trim_and_cast<T>, 0, stream, num_elements, input_width, output_width, in, out);
	}
}

template void trim_and_cast_from(hipStream_t stream, const MatrixLayout layout, const uint32_t num_elements, const uint32_t input_width, const uint32_t output_width, const float* in, float* out);
template void trim_and_cast_from(hipStream_t stream, const MatrixLayout layout, const uint32_t num_elements, const uint32_t input_width, const uint32_t output_width, const __half* in, float* out);


}
